#include <hip/hip_runtime.h> // CUDA 런타임 헤더
 // CUDA 런치 파라미터 헤더
#include <math.h> // 수학 함수 헤더

// QKV projection 커널
__global__
void qkv_projection_kernel(const float* input, const float* Wq, const float* Wk, const float* Wv,
                           float* Q, float* K, float* V,
                           int B, int T, int D) {
    int b = blockIdx.x; // b: 배치 인덱스, input/Q/K/V shape [B, T, D]
    int t = blockIdx.y; // t: 시퀀스 위치 인덱스, input/Q/K/V shape [B, T, D]
    int d = threadIdx.x; // d: 모델 차원 인덱스, input/Q/K/V shape [B, T, D]
    int idx = (b * T + t) * D + d; // idx: 1D 인덱스, maps to [b, t, d]
    float x = input[idx]; // input[b, t, d]
    float q = 0.0f; // Q[b, t, d]
    for (int k = 0; k < D; ++k) {
        q += input[(b * T + t) * D + k] * Wq[k * D + d]; // input[b, t, k] * Wq[k, d]
    }
    Q[idx] = q; // Q[b, t, d]
    float k_val = 0.0f; // K[b, t, d]
    for (int k = 0; k < D; ++k) {
        k_val += input[(b * T + t) * D + k] * Wk[k * D + d]; // input[b, t, k] * Wk[k, d]
    }
    K[idx] = k_val; // K[b, t, d]
    float v_val = 0.0f; // V[b, t, d]
    for (int k = 0; k < D; ++k) {
        v_val += input[(b * T + t) * D + k] * Wv[k * D + d]; // input[b, t, k] * Wv[k, d]
    }
    V[idx] = v_val; // V[b, t, d]
}

// Scaled Dot-Product Attention 커널
__global__
void sdpa_kernel(const float* Q, const float* K, const float* V, float* O,
                 int B, int T, int D, int H) {
    extern __shared__ float scores[]; // scores[T]
    int d_h = D / H; // 각 헤드 차원 크기
    int b = blockIdx.x; // 배치 인덱스
    int h = blockIdx.y; // 헤드 인덱스
    int t = blockIdx.z; // 시퀀스 위치 인덱스
    int dh = threadIdx.x; // 헤드 내부 차원 인덱스
    const float* Qh = Q + (b * T * D) + t * D + h * d_h; // Q[b, t, h*d_h : h*d_h+d_h]
    const float* Kh = K + (b * T * D) + 0 * D + h * d_h; // K[b, :, h*d_h : h*d_h+d_h]
    if (dh == 0) {
        for (int j = 0; j < T; ++j) {
            float dot = 0.0f; // 점곱
            for (int k = 0; k < d_h; ++k) {
                dot += Qh[k] * Kh[j * D + k]; // Q[b,t,h,k] * K[b,j,h,k]
            }
            scores[j] = dot / sqrtf((float)d_h); // 정규화
        }
        float max_score = scores[0]; // softmax 최대값
        for (int j = 1; j < T; ++j) if (scores[j] > max_score) max_score = scores[j];
        float sum_exp = 0.0f; // exp 합산
        for (int j = 0; j < T; ++j) {
            scores[j] = expf(scores[j] - max_score); // exp
            sum_exp += scores[j];
        }
        for (int j = 0; j < T; ++j) scores[j] /= sum_exp; // softmax 확률
    }
    __syncthreads();
    float out = 0.0f; // O[b, t, h, dh]
    for (int j = 0; j < T; ++j) {
        out += scores[j] * V[(b * T * D) + j * D + h * d_h + dh]; // V[b,j,h,dh]
    }
    O[(b * T * D) + t * D + h * d_h + dh] = out; // O[b,t,h,dh]
}

// Output projection 커널
__global__
void output_projection_kernel(const float* O, const float* Wo, float* output,
                              int B, int T, int D) {
    int b = blockIdx.x; // 배치 인덱스
    int t = blockIdx.y; // 시퀀스 위치 인덱스
    int d = threadIdx.x; // 모델 차원 인덱스
    int idx = (b * T + t) * D + d; // 1D 인덱스
    float sum = 0.0f; // output[b, t, d]
    for (int k = 0; k < D; ++k) {
        sum += O[b * T * D + t * D + k] * Wo[k * D + d]; // O[b,t,k] * Wo[k,d]
    }
    output[idx] = sum; // output[b, t, d]
}

// Host 함수
void multi_head_attention(const float* input, const float* Wq, const float* Wk, const float* Wv, const float* Wo,
                          float* output, int B, int T, int D, int H, hipStream_t stream = 0) {
    int size = B * T * D; // 전체 요소 수
    float *Q, *K, *V, *O; // 중간 버퍼
    hipMalloc(&Q, size * sizeof(float)); // Q: [B, T, D]
    hipMalloc(&K, size * sizeof(float)); // K: [B, T, D]
    hipMalloc(&V, size * sizeof(float)); // V: [B, T, D]
    hipMalloc(&O, size * sizeof(float)); // O: [B, T, D]
    dim3 grid1(B, T); // 그리드 크기 (B, T)
    dim3 block1(D);   // 블록 크기 (D)
    qkv_projection_kernel<<<grid1, block1, 0, stream>>>(input, Wq, Wk, Wv, Q, K, V, B, T, D);
    dim3 grid2(B, H, T); // 그리드 크기 (B, H, T)
    dim3 block2(D / H);  // 블록 크기 (D/H)
    int shared_mem = T * sizeof(float); // 공유 메모리 크기
    sdpa_kernel<<<grid2, block2, shared_mem, stream>>>(Q, K, V, O, B, T, D, H);
    dim3 grid3(B, T); // 그리드 크기 (B, T)
    dim3 block3(D);   // 블록 크기 (D)
    output_projection_kernel<<<grid3, block3, 0, stream>>>(O, Wo, output, B, T, D);
    hipFree(Q); hipFree(K); hipFree(V); hipFree(O); // 메모리 해제
}
